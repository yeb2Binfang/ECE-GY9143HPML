#include <cassert>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <assert.h>

template<typename T>
__global__ void conv2d(int batch, int color, int rows, int cols, int kCols, int kRows, T* matrix, float* kernel, T* result){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int kCenterX = kCols / 2;
	int kCenterY = kRows / 2;

	for(int b = 0; b < batch; b++){
		for(int c = 0; c < color; c++){
			for(int i = 0; i < rows;i++){
				for(int j = 0; j < cols; j++){
					for(int m = 0; m <kRows; m++){
						int mm = kRows - 1 - m;
						for(int n = 0; n < kCols; n++){
							int nn = kCols - 1 - n;

							int ii = i + (kCenterY - mm);
							int jj = j + (kCenterX - nn);

							if(ii >= 0 && ii < rows && jj >= 0 && jj < cols){
								result[b * color * rows * cols + c * rows * cols + i * cols + j] += matrix[b * c * ii * jj + c * ii * jj + ii * kRows + jj] * kernel[mm * kRows + nn];
								result[tid] = result[b * color * rows * cols + c * rows * cols + i * cols + j];
							}
						}
					}
				}
			}
		}
	}
}



void report_gpu_mem(){
	 size_t free, total;
	 hipMemGetInfo(&free, &total);
	 std::cout << "Free = " << free << " Total = " << total <<std::endl;
}

void verify_result(float* matrix, float* kernel, float* result){
	int check_sum = 0;
	int kRows = 3;
	int kCols = 3;
	int rows = 1024;
	int cols = 1024;
	int color = 3;
	int kCenterX = kCols / 2;
        int kCenterY = kRows / 2;

        for(int b = 0; b < 64; b++){
                for(int c = 0; c < 3; c++){
                        for(int i = 0; i < rows;i++){
                                for(int j = 0; j < cols; j++){
                                        for(int m = 0; m <kRows; m++){
                                                int mm = kRows - 1 - m;
                                                for(int n = 0; n < kCols; n++){
                                                        int nn = kCols - 1 - n;

                                                        int ii = i + (kCenterY - mm);
                                                        int jj = j + (kCenterX - nn);

                                                        if(ii >= 0 && ii < rows && jj >= 0 && jj < cols){
                                                                result[b * color * rows * cols + c * rows * cols + i * cols + j] += matrix[b * c * ii * jj + c * ii * jj + ii * kRows + jj] * kernel[mm * kRows + nn];
                                                                
                                                        }
                                                }
                                        }
                                }
                        }
                }
        }
	std::cout << "The check sum is" << check_sum << std::endl;
}
int main() {
	int n = 3 * 1024 * 1024;
	int size = 3 * 3 * 3;
	
	
	int bytes_n = n * sizeof(float);
	//int bytes_size = size * sizeof(float);

	float *matrix = new float[n];

	for(int i = 0; i < n; i++){
		int temp1 = i / 1024 / 1024; // channel
		int temp2 = i / 1024; //col
		int temp3 = i % 1024; //col
		matrix[i] = temp1 * (temp2 + temp3);
	}
	float *kernel = new float[size];
	for(int i = 0; i < size; i++){
		int temp1 = i / 3 / 3;
		int temp2 = i / 3;
		int temp3 = i % 3;
		kernel[i] = temp1 * (temp2 + temp3);
	}	

	float *result = new float[n];

	float *d_matrix, *d_result;
	hipMalloc(&d_matrix, bytes_n);
  	hipMalloc(&d_result, bytes_n);

	hipMemcpy(d_matrix, matrix, bytes_n, hipMemcpyHostToDevice);
	conv2d<<<1, 1>>>(64, 3, 1024, 1024, 3, 3, matrix, kernel, result);
	hipDeviceSynchronize();
	verify_result(matrix, kernel, result);
	delete[] matrix;
 	delete[] kernel;
  	delete[] result;
	return 0;
}
